#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cudaFunctions.cuh"
#include <iostream>
#include "opencv2\opencv.hpp"
#include "opencv2\core.hpp"

__global__ void calSqDiffKernal(float* source, float* tmplate, float* result, float* tmplateMask, float* srcMask, int rlt_width, int rlt_height, int mw, int mh, int mchannel){
	int x_ = blockIdx.x * blockDim.x + threadIdx.x;
	int y_ = blockIdx.y * blockDim.y + threadIdx.y;

	if (x_ >= rlt_width || y_ >= rlt_height)
	{
		return;
	}

	int totalPixels = mw * mh * mchannel;
	int cnt = 0;
	float re = 0.0f;
	for (int tx = 0; tx < mw; tx++)
	{
		for (int ty = 0; ty < mh; ty++)
		{
			for (int channel = 0; channel < mchannel; channel++)
			{
				if (tmplateMask[ty * mw * mchannel + tx * mchannel + channel] != 0 && srcMask[(y_ + ty) * (rlt_width + mw - 1) *mchannel + (x_ + tx)*mchannel + channel] != 0)
				{
					float temp = tmplate[ty * mw * mchannel + tx * mchannel + channel] - source[(y_ + ty) * (rlt_width + mw - 1) * mchannel + (x_ + tx)*mchannel + channel];
					re += temp * temp;
					cnt++;
				}
			}

		}
	}
	if (cnt == 0)
	{
		re = 1.1f * totalPixels;//result above max to be 1
	}
	else
	{
		re = re / (float)cnt * totalPixels;
	}

	result[y_ * (rlt_width) + x_] = re;
}

__global__ void testKernal(float* source, float* tmplate, float* result, float* tmplateMask, float* srcMask, int rlt_width, int rlt_height, int mw, int mh, int mchannel){
	int x_ = blockIdx.x * blockDim.x + threadIdx.x;
	int y_ = blockIdx.y * blockDim.y + threadIdx.y;

	if (x_ >= rlt_width || y_ >= rlt_height)
	{
		return;
	}
	result[y_ * (rlt_width)+x_] = source[(y_) * (rlt_width + mw - 1) * mchannel + (x_ )*mchannel + 0];
}

void printDevProp(hipDeviceProp_t devProp)
{
	printf("Major revision number:         %d\n", devProp.major);
	printf("Minor revision number:         %d\n", devProp.minor);
	printf("Name:                          %s\n", devProp.name);
	printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
	printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Maximum memory pitch:          %lu\n", devProp.memPitch);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n", devProp.clockRate);
	printf("Total constant memory:         %lu\n", devProp.totalConstMem);
	printf("Texture alignment:             %lu\n", devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}

void DIYmatchTemplate(const cv::Mat& source, const cv::Mat& tmplate, cv::Mat& result, const cv::Mat& tmplateMask, const cv::Mat& srcMask){
	//just use cv_tm_sqdiff method
	assert(source.size() == srcMask.size() && source.channels() == srcMask.channels());
	assert(tmplate.size() == tmplateMask.size() && tmplate.channels() == tmplateMask.channels());
	assert(tmplate.type() == source.type());
	
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}
#if 0
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printDevProp(devProp);
#endif
	//copydata !the memory may not be continuous in Mat ! Mat MUST be row major!
	const uchar* ptr;

	float* src = nullptr;
	cudaStatus = hipMalloc((void**)&src, source.rows * source.cols * source.channels() * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	for (int i = 0; i < source.rows; i++)
	{
		ptr = source.ptr(i);
		float *src_ptr = src + (source.cols * source.channels()) * i;
		cudaStatus = hipMemcpy(src_ptr, (void*)ptr, source.cols * source.channels() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "cudaCopy failed!");
	}

	float* tmpl = nullptr;
	cudaStatus = hipMalloc((void**)&tmpl, tmplate.rows * tmplate.cols * tmplate.channels() * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	for (int i = 0; i < tmplate.rows; i++)
	{
		ptr = tmplate.ptr(i);
		float *tmpl_ptr = tmpl + (tmplate.cols * tmplate.channels()) * i;
		cudaStatus = hipMemcpy(tmpl_ptr, (void*)ptr, tmplate.cols * tmplate.channels() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "cudaCopy failed!");
	}

	float* rlt = nullptr;
	ptr = result.ptr(0);
	cudaStatus = hipMalloc((void**)&rlt, result.rows * result.cols * result.channels() * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	float* tmplM = nullptr;
	cudaStatus = hipMalloc((void**)&tmplM, tmplateMask.rows * tmplateMask.cols * tmplateMask.channels() * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	for (int i = 0; i < tmplateMask.rows; i++)
	{
		ptr = tmplateMask.ptr(i);
		float *tmplM_ptr = tmplM + (tmplateMask.cols * tmplateMask.channels()) * i;
		cudaStatus = hipMemcpy(tmplM_ptr, (void*)ptr, tmplateMask.cols * tmplateMask.channels() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "cudaCopy failed!");
	}

	float* srcM = nullptr;
	cudaStatus = hipMalloc((void**)&srcM, srcMask.rows * srcMask.cols * srcMask.channels() * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");
	for (int i = 0; i < srcMask.rows; i++)
	{
		ptr = srcMask.ptr(i);
		float *srcM_ptr = srcM + (srcMask.cols * srcMask.channels()) * i;
		cudaStatus = hipMemcpy(srcM_ptr, (void*)ptr, srcMask.cols * srcMask.channels() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "cudaCopy failed!");
	}
	

	int width = result.cols;
	int height = result.rows;
	int maskWidth = tmplate.cols;
	int maskHeight = tmplate.rows;
	int maskCh = tmplate.channels();

	//run kernal
	dim3 blocks(result.cols / 16 + 1, result.rows / 16 + 1);
	dim3 threads(16, 16);
	calSqDiffKernal << <blocks, threads >> >(src, tmpl, rlt, tmplM, srcM, width, height, maskWidth, maskHeight, maskCh);
	//testKernal << <blocks, threads >> >(src, tmpl, rlt, tmplM, srcM, width, height, maskWidth, maskHeight, maskCh);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	//copy back
	for (int i = 0; i < result.rows; i++)
	{
		ptr = result.ptr(i);
		float *rlt_ptr = rlt + (result.cols * result.channels()) * i;
		cudaStatus = hipMemcpy((void*)ptr, rlt_ptr, result.cols * result.channels() * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "cudaCopyBack failed!");
	}
	hipFree(src);
	hipFree(tmpl);
	hipFree(rlt);
	hipFree(tmplM); 
	hipFree(srcM);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	printf("%d", i);
}

void addVector(int *c, const int *b, const int *a, const int size){
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.  
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .  
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	addKernel <<<1, size >>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel  
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns  
	// any errors encountered during the launch.  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.  
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}


	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}